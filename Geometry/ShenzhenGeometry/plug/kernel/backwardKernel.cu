#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/hip/hip_vector_types.h"
#include "../include/helper_geometry.h"

#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16
#define PI 3.14159265359
#define CHECK_CUDA(x) AT_ASSERTM(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) AT_ASSERTM(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

// 弦图的纹理内存
texture<float, hipTextureType3D, hipReadModeElementType> sinoTexture;

__global__ void backwardKernel(float* volume, const uint3 volumeSize, const uint2 detectorSize, const float* projectVector, const uint index,const int anglesNum,const float3 volumeCenter, const float2 detectorCenter){
    // 体素驱动，代表一个体素点
   uint2 volumeIdx = make_uint2(blockIdx.x*blockDim.x + threadIdx.x, blockIdx.y*blockDim.y + threadIdx.y);
   if(volumeIdx.x >= volumeSize.x  || volumeIdx.y >= volumeSize.y )
      return;
   float3 sourcePosition = make_float3(projectVector[0], projectVector[1], projectVector[2]);
   float3 detectorPosition = make_float3(projectVector[3], projectVector[4], projectVector[5]);
   float3 u = make_float3(projectVector[6], projectVector[7], projectVector[8]);
   float3 v = make_float3(projectVector[9], projectVector[10], projectVector[11]);
   float sampleInterval = fabs(sourcePosition.z)/fabs(sourcePosition.z-detectorPosition.z);
   for (int k=0;k<volumeSize.z;k++)
   {
        const float3 coordinates = make_float3(volumeCenter.x + volumeIdx.x, volumeCenter.y + volumeIdx.y,volumeCenter.z+k);
        float3 normVector=cross(u,v);
        float3 intersection=intersectLines3D(sourcePosition,coordinates,detectorPosition,detectorPosition+normVector);
        float detectorX=dot(intersection-detectorPosition,u)-detectorCenter.x;
        float detectorY=dot(intersection-detectorPosition,v)-detectorCenter.y;
        int idx = k * volumeSize.x * volumeSize.y + volumeIdx.y * volumeSize.x + volumeIdx.x;
        float val = tex3D(sinoTexture, detectorX + 0.5f, detectorY + 0.5f, index+0.5f);
        volume[idx] += val * 2*PI / anglesNum;
   }
/*
    // 计算得到探测器像素坐标x,z
   float sampleInterval = sid / sdd;
   const float2 coordinates = make_float2(volumeCenter.x + volumeIdx.x, volumeCenter.y + volumeIdx.y) * sampleInterval;

   angle = angle / 180 * PI - PI / 2;
   float2 ex = make_float2(cos(angle), sin(angle));
   float2 ey = make_float2(-ex.y, ex.x);
   float2 source = ex * sid;
   float2 detector = ex * (sdd - sid);
   float2 intersection = intersectLines2D(coordinates, -1*source, detector, detector + ey);
   float x = dot(intersection, ey) - detectorCenter.x;

   float2 biasRay = source + coordinates;
   float dz = sdd / dot(biasRay, ex) * sampleInterval;
   float z = volumeCenter.z * dz - detectorCenter.y;

    // 计算权重
   float coff = sid / (sid - dot(coordinates, ex));

    // 反投影
   for (int k = 0; k < volumeSize.z; k++){
       int idx = k * volumeSize.x * volumeSize.y + volumeIdx.y * volumeSize.x + volumeIdx.x;
       float val = tex3D(sinoTexture, x + 0.5f, z + 0.5f, index+0.5f);
       volume[idx] += val * coff * coff * PI / anglesNum;
       z += dz;
   }*/
}

torch::Tensor backward(torch::Tensor sino, torch::Tensor _volumeSize, torch::Tensor _detectorSize, torch::Tensor projectVector, const long device){
    CHECK_INPUT(sino);
    CHECK_INPUT(_volumeSize);
    AT_ASSERTM(_volumeSize.size(0) == 3, "volume size's length must be 3");
    CHECK_INPUT(_detectorSize);
    AT_ASSERTM(_detectorSize.size(0) == 2, "detector size's length must be 2");
    CHECK_INPUT(projectVector);
    AT_ASSERTM(projectVector.size(1) == 12, "project vector's shape must be [angle's number, 12]");

    int angles = projectVector.size(0);
    auto out = torch::zeros({sino.size(0), _volumeSize[2].item<int>(), _volumeSize[1].item<int>(), _volumeSize[0].item<int>()}).to(sino.device());
    float* outPtr = out.data<float>();
    float* sinoPtr = sino.data<float>();

    // 初始化纹理
    hipSetDevice(device);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    sinoTexture.addressMode[0] = hipAddressModeBorder;
    sinoTexture.addressMode[1] = hipAddressModeBorder;
    sinoTexture.addressMode[2] = hipAddressModeBorder;
    sinoTexture.filterMode = hipFilterModeLinear;
    sinoTexture.normalized = false;

    // 体块和探测器的大小位置向量化
    uint3 volumeSize = make_uint3(_volumeSize[0].item<int>(), _volumeSize[1].item<int>(), _volumeSize[2].item<int>());
    uint2 detectorSize = make_uint2(_detectorSize[0].item<int>(), _detectorSize[1].item<int>());
    float3 volumeCenter = make_float3(volumeSize) / -2.0;
    float2 detectorCenter = make_float2(detectorSize) / -2.0;
    for(int batch = 0;batch < sino.size(0); batch++){
        float* sinoPtrPitch = sinoPtr + detectorSize.x * detectorSize.y * angles * batch;
        float* outPtrPitch = outPtr + volumeSize.x * volumeSize.y * volumeSize.z * batch;

        // 绑定纹理
        hipExtent m_extent = make_hipExtent(detectorSize.x, detectorSize.y, angles);
        hipArray *sinoArray;
        hipMalloc3DArray(&sinoArray, &channelDesc, m_extent);
        hipMemcpy3DParms copyParams = {0};
        copyParams.srcPtr = make_hipPitchedPtr((void*)sinoPtrPitch, detectorSize.x*sizeof(float), detectorSize.x, detectorSize.y);
        copyParams.dstArray = sinoArray;
        copyParams.kind = hipMemcpyDeviceToDevice;
        copyParams.extent = m_extent;
        hipMemcpy3D(&copyParams);
        hipBindTextureToArray(sinoTexture, sinoArray, channelDesc);

        // 以角度为单位做体素驱动的反投影
        const dim3 blockSize = dim3(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
        const dim3 gridSize = dim3(volumeSize.x / blockSize.x + 1, volumeSize.y / blockSize.y + 1 , 1);
        for (int angle = 0; angle < angles; angle++){
           backwardKernel<<<gridSize, blockSize>>>(outPtrPitch, volumeSize, detectorSize, (float*)projectVector[angle].data<float>(), angle,angles,volumeCenter,detectorCenter);
        }

      // 解绑纹理
      hipUnbindTexture(sinoTexture);
      hipFreeArray(sinoArray);
    }
    return out;
}